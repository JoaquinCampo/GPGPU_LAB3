/**
 * @file main.cu
 * @brief Kernel and host code for naive GPU matrix transpose using global memory.
 *
 * Implements a __global__ kernel that transposes an integer matrix on the GPU
 * by reading and writing only from/to global memory. The host-side code
 * allocates buffers, initializes data, launches the kernel, measures execution time
 * over multiple runs, and verifies correctness of the result.
 *
 * Usage:
 *   ./programa [rows cols]
 * Default matrix size is 1024x1024 if no arguments are provided.
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>

/**
 * @brief Naive transpose kernel using only global memory.
 *
 * Each thread computes its 2D coordinates and performs the transpose
 * by reading from input at (row, col) and writing to output at (col, row).
 * No shared memory or tiling used: serves as baseline for memory-access analysis.
 *
 * @param in    Pointer to input matrix in row-major order (rows x cols).
 * @param out   Pointer to output matrix in row-major order (cols x rows).
 * @param rows  Number of rows in the input matrix.
 * @param cols  Number of columns in the input matrix.
 */
__global__ void transposeNaive(const int* in, int* out, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        // Transpose element
        out[col * rows + row] = in[row * cols + col];
    }
}

/**
 * @brief Host entry point: sets up data, launches kernel, measures time, validates.
 *
 * - Parses optional command-line arguments for matrix dimensions.
 * - Allocates host and device buffers, initializes input with sequential values.
 * - Launches kernel with 32x32 thread blocks and grid covering the matrix.
 * - Performs a warm-up run, then times 10 executions using CUDA events.
 * - Calculates average and standard deviation of execution times.
 * - Copies result back to host and verifies correctness element-wise.
 *
 * @param argc  Number of command-line arguments.
 * @param argv  Array of argument strings (rows, cols).
 * @return      Returns 0 on success.
 */
int main(int argc, char* argv[]) {
    // Matrix dimensions (default 1024x1024)
    int rows = 1024;
    int cols = 1024;
    if (argc >= 3) {
        rows = std::atoi(argv[1]);
        cols = std::atoi(argv[2]);
    }
    std::cout << "Matrix size: " << rows << " x " << cols << std::endl;

    size_t size = static_cast<size_t>(rows) * cols;
    size_t bytes = size * sizeof(int);

    // Host allocations
    std::vector<int> h_in(size);
    std::vector<int> h_out(size);

    // Initialize input matrix
    for (size_t i = 0; i < size; ++i) {
        h_in[i] = static_cast<int>(i);
    }

    // Device allocations
    int *d_in = nullptr, *d_out = nullptr;
    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, bytes);

    hipMemcpy(d_in, h_in.data(), bytes, hipMemcpyHostToDevice);

    // Kernel configuration
    dim3 blockDim(32, 32);
    dim3 gridDim((cols + blockDim.x - 1) / blockDim.x,
                 (rows + blockDim.y - 1) / blockDim.y);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int iterations = 10;
    std::vector<float> times(iterations);

    // Warm-up
    transposeNaive<<<gridDim, blockDim>>>(d_in, d_out, rows, cols);
    hipDeviceSynchronize();

    // Timed runs
    for (int i = 0; i < iterations; ++i) {
        hipEventRecord(start);
        transposeNaive<<<gridDim, blockDim>>>(d_in, d_out, rows, cols);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms = 0.0f;
        hipEventElapsedTime(&ms, start, stop);
        times[i] = ms;
    }

    // Compute average and standard deviation
    float sum = 0.0f;
    for (float t : times) sum += t;
    float avg = sum / iterations;
    float sq_sum = 0.0f;
    for (float t : times) sq_sum += (t - avg) * (t - avg);
    float stddev = std::sqrt(sq_sum / iterations);

    std::cout << "Average time over " << iterations
              << " runs: " << avg << " ms (± " << stddev << " ms)" << std::endl;

    // Cleanup
    hipFree(d_in);
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
} 