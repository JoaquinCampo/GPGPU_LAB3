/**
 * @file ej1_part1.cu
 * @brief Kernel y código host para transposición de matriz usando memoria global en GPU.
 */

// Inclusión de bibliotecas necesarias
#include <hip/hip_runtime.h>  // Para funciones de CUDA
#include <iostream>        // Para operaciones de entrada/salida
#include <vector>         // Para manejo de vectores
#include <cmath>          // Para funciones matemáticas
#include <nvtx3/nvToolsExt.h>  // Para profiling

// Constantes del programa
#define MAX_DIM 4096  // Dimensión máxima de la matriz

// Macro para verificar errores de CUDA
#define CUDA_CHK(ans) do { gpuAssert((ans), __FILE__, __LINE__); } while(0)

// Función para manejar errores de CUDA
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
 * @brief Kernel de transposición naive usando solo memoria global.
 *
 * Cada thread calcula sus coordenadas 2D y realiza la transposición
 * leyendo desde la entrada en (fila, columna) y escribiendo en la salida en (columna, fila).
 * No se usa memoria compartida ni tiling: sirve como línea base para análisis de acceso a memoria.
 *
 * @param in    Puntero a matriz de entrada en orden fila-mayor (filas x columnas)
 * @param out   Puntero a matriz de salida en orden fila-mayor (columnas x filas)
 * @param rows  Número de filas en la matriz de entrada
 * @param cols  Número de columnas en la matriz de entrada
 */
__global__ void transposeNaive(const int *in, int *out, int rows, int cols)
{
    // Calcula las coordenadas 2D del thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Verifica que el thread esté dentro de los límites de la matriz
    if (row < rows && col < cols)
    {
        // Realiza la transposición del elemento
        out[col * rows + row] = in[row * cols + col];
    }
}

/**
 * @brief Punto de entrada del host para transposición de matriz en GPU.
 *
 * @param argc  Número de argumentos de línea de comando
 * @param argv  Array de strings de argumentos (opcionalmente: filas, columnas, blockX, blockY)
 * @return      0 si la transposición es correcta, 1 para error de uso/dimensión, 2 para verificación fallida
 */
int main(int argc, char *argv[]) 
{
    // 1) Parsear argumentos
    int rows = 1024, cols = 1024;
    int blockX = 32, blockY = 32;  // Dimensiones de bloque por defecto
    
    // Parsear argumentos de línea de comando
    if (argc == 3) {
        rows = std::atoi(argv[1]);
        cols = std::atoi(argv[2]);
    } else if (argc == 5) {
        rows = std::atoi(argv[1]);
        cols = std::atoi(argv[2]);
        blockX = std::atoi(argv[3]);
        blockY = std::atoi(argv[4]);
    } else if (argc != 1) {
        std::cerr << "Usage: " << argv[0] << " [rows cols [blockX blockY]]\n";
        return 1;
    }

    // Validar dimensiones
    if (rows > MAX_DIM || cols > MAX_DIM) {
        std::cerr << "Error: Matrix dimensions must be ≤ " << MAX_DIM << "\n";
        return 1;
    }
    if (blockX <= 0 || blockY <= 0 || blockX * blockY > 1024) {
        std::cerr << "Error: Invalid block dimensions (" << blockX << "x" << blockY
                  << "). Product must be > 0 and <= 1024.\n";
        return 1;
    }

    std::cout << "Matrix size: " << rows << " x " << cols
              << ", Block size: " << blockX << " x " << blockY << "\n";

    // 2) Asignar e inicializar arrays en host
    size_t size = static_cast<size_t>(rows) * cols;
    size_t bytes = size * sizeof(int);
    std::vector<int> h_in(size);
    std::vector<int> h_out(size);

    nvtxRangePushA("Init in");
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            h_in[i * cols + j] = i * cols + j;
        }
    }
    nvtxRangePop();

    // 3) Asignar arrays en dispositivo
    int *d_in = nullptr, *d_out = nullptr;
    nvtxRangePushA("Malloc in");
    CUDA_CHK(hipMalloc(&d_in, bytes));
    nvtxRangePop();

    nvtxRangePushA("Malloc out");
    CUDA_CHK(hipMalloc(&d_out, bytes));
    nvtxRangePop();

    nvtxRangePushA("H2D memcpy");
    CUDA_CHK(hipMemcpy(d_in, h_in.data(), bytes, hipMemcpyHostToDevice));
    nvtxRangePop();

    // 4) Configuración del lanzamiento del kernel
    dim3 blockDim(blockX, blockY);
    
    // Calcular dimensiones de la grilla
    int remainder_x = cols % blockDim.x;
    int remainder_y = rows % blockDim.y;
    int numBlocksX = cols / blockDim.x + (remainder_x > 0 ? 1 : 0);
    int numBlocksY = rows / blockDim.y + (remainder_y > 0 ? 1 : 0);
    dim3 gridDim(numBlocksX, numBlocksY);

    // 5) Lanzar kernel
    nvtxRangePushA("Kernel launch");
    transposeNaive<<<gridDim, blockDim>>>(d_in, d_out, rows, cols);
    CUDA_CHK(hipGetLastError());
    CUDA_CHK(hipDeviceSynchronize());
    nvtxRangePop();

    // 6) Copiar de vuelta y verificar corrección
    nvtxRangePushA("D2H memcpy");
    CUDA_CHK(hipMemcpy(h_out.data(), d_out, bytes, hipMemcpyDeviceToHost));
    nvtxRangePop();

    // Verificar resultados
    bool ok = true;
    for (int r = 0; r < rows && ok; ++r) {
        for (int c = 0; c < cols; ++c) {
            int expected = r * cols + c;
            if (h_out[c * rows + r] != expected) {
                std::cerr << "FAILED at (" << r << "," << c << "): "
                          << h_out[c * rows + r] << " != " << expected << "\n";
                ok = false;
                break;
            }
        }
    }
    std::cout << (ok ? "Transpose OK\n" : "Transpose FAILED\n");

    // 7) Limpieza
    CUDA_CHK(hipFree(d_in));
    CUDA_CHK(hipFree(d_out));
    return ok ? 0 : 2;
}