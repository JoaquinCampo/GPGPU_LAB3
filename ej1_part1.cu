/**
 * @file main.cu
 * @brief kernel and host code for naive gpu matrix transpose using global memory.
 *
 * implements a __global__ kernel that transposes an integer matrix on the gpu
 * by reading and writing only from/to global memory. the host-side code
 * allocates buffers, initializes data, launches the kernel, measures execution time
 * over multiple runs, and verifies correctness of the result.
 *
 * usage:
 *   ./programa [rows cols]
 * default matrix size is 1024x1024 if no arguments are provided.
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <nvtx3/nvToolsExt.h>


#define MAX_DIM 4096

#define CUDA_CHK(ans) do { gpuAssert((ans), __FILE__, __LINE__); } while(0)
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"gpuAssert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/**
 * @brief naive transpose kernel using only global memory.
 *
 * each thread computes its 2d coordinates and performs the transpose
 * by reading from input at (row, col) and writing to output at (col, row).
 * no shared memory or tiling used: serves as baseline for memory-access analysis.
 *
 * @param in    pointer to input matrix in row-major order (rows x cols).
 * @param out   pointer to output matrix in row-major order (cols x rows).
 * @param rows  number of rows in the input matrix.
 * @param cols  number of columns in the input matrix.
 */
__global__ void transposeNaive(const int *in, int *out, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols)
    {
        // transpose element
        out[col * rows + row] = in[row * cols + col];
    }
}


/**
 * @brief host entry point for naive gpu matrix transpose (global memory only).
 *
 * steps performed:
 * - parses optional command-line arguments for matrix dimensions (rows, cols).
 * - allocates and initializes host input matrix with sequential values.
 * - allocates device memory for input and output matrices.
 * - copies input data to device.
 * - configures and launches the transposenaive kernel (32x32 thread blocks).
 * - synchronizes and checks for kernel errors.
 * - copies the transposed result back to host.
 * - verifies correctness by comparing each element to the expected value.
 * - prints whether the transpose succeeded or failed.
 * - frees device memory before exit.
 *
 * usage:
 *   ./programa [rows cols]
 *   (defaults: rows=1024, cols=1024)
 *
 * @param argc  number of command-line arguments.
 * @param argv  array of argument strings (optionally: rows, cols).
 * @return      0 if transpose is correct, 1 for usage/dimension error, 2 for failed verification.
 */
int main(int argc, char *argv[]) {
    // 1) Parse arguments
    int rows = 1024, cols = 1024;
    int blockX = 32, blockY = 32; // Default block dimensions
    // Use 1D vectors for host arrays to avoid stack overflow
    // A REVISAR - CHEQUEAR SI ESTA IMPLEMENTACION DE MATRICES ESTA BIEN, O SI TENEMOS QUE USAR [] []. CUANDO USO [] [] ME TIRA ERROR DE EJECUCIÓN POR ALOCACIÓN DE MEMORIA.
    std::vector<int> h_in(rows * cols);
    std::vector<int> h_out(rows * cols);
    if (argc == 3) {
        rows = std::atoi(argv[1]);
        cols = std::atoi(argv[2]);
    } else if (argc == 5) {
        rows = std::atoi(argv[1]);
        cols = std::atoi(argv[2]);
        blockX = std::atoi(argv[3]);
        blockY = std::atoi(argv[4]);
    } else if (argc != 1) {
        std::cerr << "Usage: " << argv[0] << " [rows cols [blockX blockY]]\n";
        return 1;
    }

    if (rows > MAX_DIM || cols > MAX_DIM) {
        std::cerr << "Error: dims must be ≤ " << MAX_DIM << "\n";
        return 1;
    }
    if (blockX <= 0 || blockY <= 0 || blockX * blockY > 1024) { // Check block size validity
         std::cerr << "Error: Invalid block dimensions (" << blockX << "x" << blockY
                   << "). Product must be > 0 and <= 1024.\n";
         return 1;
    }

    std::cout << "Matrix size: " << rows << " x " << cols
              << ", Block size: " << blockX << " x " << blockY << "\n";

    size_t size = static_cast<size_t>(rows) * cols;
    size_t bytes = size * sizeof(int);

    // 2) Allocate & init host arrays
    nvtxRangePushA("Init in");
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            h_in[i * cols + j] = i * cols + j;
        }
    }
    nvtxRangePop();

    // 3) Allocate device arrays
    int *d_in = nullptr, *d_out = nullptr;
    nvtxRangePushA("Malloc in");
    std::cout << "[DEBUG] Allocating d_in with cudaMalloc, bytes: " << bytes << std::endl;
    hipError_t err_in = hipMalloc(&d_in,  bytes);
    if (err_in != hipSuccess) {
        std::cerr << "[ERROR] hipMalloc for d_in failed: " << hipGetErrorString(err_in) << std::endl;
        return 1;
    }
    nvtxRangePop();
    nvtxRangePushA("Malloc out");
    std::cout << "[DEBUG] Allocating d_out with cudaMalloc, bytes: " << bytes << std::endl;
    hipError_t err_out = hipMalloc(&d_out, bytes);
    if (err_out != hipSuccess) {
        std::cerr << "[ERROR] hipMalloc for d_out failed: " << hipGetErrorString(err_out) << std::endl;
        hipFree(d_in);
        return 1;
    }
    nvtxRangePop();

    nvtxRangePushA("H2D memcpy");
        CUDA_CHK(hipMemcpy(d_in, h_in.data(), bytes, hipMemcpyHostToDevice));
    nvtxRangePop();

    // 4) Kernel launch config
    dim3 blockDim(blockX, blockY); // Use parsed block dimensions

    // Calculate how many blocks are needed in each dimension
    int remainder_x = cols % blockDim.x;
    int remainder_y = rows % blockDim.y;

    // If there is a remainder, we need one extra block to cover the edge
    int numBlocksX = cols / blockDim.x + (remainder_x > 0 ? 1 : 0);
    int numBlocksY = rows / blockDim.y + (remainder_y > 0 ? 1 : 0);
    dim3 gridDim(numBlocksX, numBlocksY);

    // 5) Launch once
    nvtxRangePushA("Kernel launch");
        transposeNaive<<<gridDim, blockDim>>>(d_in, d_out, rows, cols);
        CUDA_CHK(hipGetLastError());
        CUDA_CHK(hipDeviceSynchronize());
    nvtxRangePop();

    // 6) Copy back & verify correctness
    nvtxRangePushA("D2H memcpy");
        CUDA_CHK(hipMemcpy(h_out.data(), d_out, bytes, hipMemcpyDeviceToHost));
    nvtxRangePop();

    bool ok = true;
    for (int r = 0; r < rows && ok; ++r) {
      for (int c = 0; c < cols; ++c) {
        int expected = r * cols + c;
        if (h_out[c * rows + r] != expected) {
            std::cerr << "FAILED at ("<<r<<","<<c<<"): " << h_out[c * rows + r] << " != " << expected << "\n";
            ok = false;
            break;
        }
      }
    }
    std::cout << (ok ? "Transpose OK\n" : "Transpose FAILED\n");

    // 7) Cleanup
    CUDA_CHK(hipFree(d_in));
    CUDA_CHK(hipFree(d_out));
    return ok ? 0 : 2;
}