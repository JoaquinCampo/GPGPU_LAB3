#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <nvtx3/nvToolsExt.h>

#define MAX_ROWS 4096
#define MAX_COLS 4096

#define CUDA_CHK(ans) do { gpuAssert((ans), __FILE__, __LINE__); } while(0)
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"gpuAssert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void transposeSharedNoPad(const int* in, int* out, int rows, int cols) {
    extern __shared__ int tile[];
    int blockX = blockDim.x;
    int blockY = blockDim.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * blockY + ty;
    int col = blockIdx.x * blockX + tx;

    if (row < rows && col < cols) {
        tile[ty * blockX + tx] = in[row * cols + col];
    }
    __syncthreads();

    int trow = blockIdx.x * blockX + ty;
    int tcol = blockIdx.y * blockY + tx;
    if (trow < cols && tcol < rows) {
        out[trow * rows + tcol] = tile[tx * blockX + ty];
    }
}

int main(int argc, char* argv[]) {
    int rows = 1024, cols = 1024;
    int blockX = 32, blockY = 32; 
    std::vector<int> h_in(rows * cols);
    std::vector<int> h_out(rows * cols);

    rows = std::atoi(argv[1]);
    cols = std::atoi(argv[2]);
    blockX = std::atoi(argv[3]);
    blockY = std::atoi(argv[4]);

    std::cout << "Matrix size: " << rows << " x " << cols << ", Block size: " << blockX << " x " << blockY << "\n";

    size_t size = static_cast<size_t>(rows) * cols;
    size_t bytes = size * sizeof(int);

    nvtxRangePushA("Init in");
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            h_in[i * cols + j] = i * cols + j;
        }
    }
    nvtxRangePop();

    int *d_in = nullptr, *d_out = nullptr;
    
    nvtxRangePushA("Malloc in");
    hipError_t err_in = hipMalloc(&d_in,  bytes);
    if (err_in != hipSuccess) {
        std::cerr << "[ERROR] hipMalloc for d_in failed: " << hipGetErrorString(err_in) << std::endl;
        return 1;
    }
    nvtxRangePop();

    nvtxRangePushA("Malloc out");
    hipError_t err_out = hipMalloc(&d_out, bytes);
    if (err_out != hipSuccess) {
        std::cerr << "[ERROR] hipMalloc for d_out failed: " << hipGetErrorString(err_out) << std::endl;
        hipFree(d_in);
        return 1;
    }
    nvtxRangePop();

    nvtxRangePushA("H2D memcpy");
        CUDA_CHK(hipMemcpy(d_in, h_in.data(), bytes, hipMemcpyHostToDevice));
    nvtxRangePop();


    dim3 blockDim(blockX, blockY); 

    int remainder_x = cols % blockDim.x;
    int remainder_y = rows % blockDim.y;

    int numBlocksX = cols / blockDim.x + (remainder_x > 0 ? 1 : 0);
    int numBlocksY = rows / blockDim.y + (remainder_y > 0 ? 1 : 0);
    dim3 gridDim(numBlocksX, numBlocksY);

    size_t sbytes = blockX * blockY * sizeof(int);
    nvtxRangePushA("Kernel launch");
        transposeSharedNoPad<<<gridDim, blockDim, sbytes>>>(d_in, d_out, rows, cols);
        CUDA_CHK(hipGetLastError());
        CUDA_CHK(hipDeviceSynchronize());
    nvtxRangePop();

    nvtxRangePushA("D2H memcpy");
        CUDA_CHK(hipMemcpy(h_out.data(), d_out, bytes, hipMemcpyDeviceToHost));
    nvtxRangePop();

    bool ok = true;
    for (int r = 0; r < rows && ok; ++r) {
        for (int c = 0; c < cols; ++c) {
            if (h_out[c * rows + r] != h_in[r * cols + c]) {
                ok = false;
                break;
            }
        }
    }
    std::cout << (ok ? "Transpose OK\n" : "Transpose not OK\n");

    CUDA_CHK(hipFree(d_in));
    CUDA_CHK(hipFree(d_out));
    return ok ? 0 : 2;
} 
